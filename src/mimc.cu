#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <gmpxx.h>
#include "cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/gpu_support.h"

#include "mimc_constants.h"
#include "mimc.h"


// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 8
#define BITS 256
#define DEFAULT_TPB 128
#define MAX_CUDA_OUT 4096
#define MAX_IPB 256 // MAX_TPB / MIN_tpi 1024/4

typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;
typedef cgbn_mem_t<BITS> bn_mem_t;
typedef typename env_t::cgbn_t bn_t;

template<typename T>
struct feistel_state_t{
    T k;
    T l;
    T r;
} ;

typedef struct {
    int64_t x;
    int64_t y;
    uint32_t side_length;
    uint32_t key;
    uint32_t rarity;
} explore_in_t;

typedef struct {
    bn_mem_t hash;
    int64_t x;
    int64_t y;
} explore_out_item_t;

typedef struct {
    explore_out_item_t planets[MAX_CUDA_OUT];
    uint32_t count;
} explore_out_t;


namespace {

    int32_t g_tpb = DEFAULT_TPB;
    __constant__ bn_mem_t g_device_p;
    __constant__ bn_mem_t g_device_c[MimcConstants::rounds];
}

__device__ void add_mod(env_t &bn_env, bn_t &r, const bn_t &a, const bn_t &b, const bn_t &modulus)
{
    cgbn_add(bn_env, r, a, b);
    if (cgbn_compare(bn_env, r, modulus) == 1) {
        cgbn_sub(bn_env, r, r, modulus);
    }
}

__device__ void mix(env_t &bn_env, feistel_state_t<bn_t> &state)
{
    bn_t bn_t_5, p, t, ci;
    cgbn_load(bn_env, p, &g_device_p);
    cgbn_set_ui32(bn_env, bn_t_5, 5);

    for (int32_t i = 0; i < MimcConstants::rounds - 1; ++i) {
        cgbn_load(bn_env, ci, g_device_c + i);
        add_mod(bn_env, t, state.l, state.k, p);
        add_mod(bn_env, t, t, ci, p);
        cgbn_modular_power(bn_env, t, t, bn_t_5, p);

        add_mod(bn_env, t, t, state.r, p);
        cgbn_set(bn_env, state.r, state.l);
        cgbn_set(bn_env, state.l, t);
    }
    add_mod(bn_env, t, state.l, state.k, p);
    cgbn_modular_power(bn_env, t, t, bn_t_5, p);
    add_mod(bn_env, state.r, t, state.r, p);
}

__device__  void inject(env_t &bn_env, feistel_state_t<bn_t> &state, bn_t elt)
{
    bn_t p;
    cgbn_load(bn_env, p, &g_device_p);
    add_mod(bn_env, state.l, state.l, elt, p);
}

__device__ void coords_to_bn(env_t &bn_env, bn_t &r, int64_t num)
{
#ifdef DF_INT32_COORDS
    if (num > 0) {
        cgbn_set_ui32(bn_env, r, num);
        return;
    }
    bn_t p;
    cgbn_load(bn_env, p, &g_device_p);
    cgbn_sub_ui32(bn_env, r, p, abs(num));
#else
    uint32_t low = llabs(num) & 0xffffffff;
    uint32_t high = llabs(num) >> 32;
    cgbn_set_ui32(bn_env, r, 0);
    cgbn_insert_bits_ui32(bn_env, r, r, 0, 32, low);
    cgbn_insert_bits_ui32(bn_env, r, r, 32, 32, high);
    if (num < 0) {
        bn_t p;
        cgbn_load(bn_env, p, &g_device_p);
        cgbn_sub(bn_env, r, p, r);
    }
#endif //DF_INT32_COORDS
}

__device__  bool is_planet(env_t &bn_env, const bn_t &hash, uint32_t rarity)
{
    bn_t threshold, p;
    cgbn_load(bn_env, p, &g_device_p);
    cgbn_div_ui32(bn_env, threshold, p, rarity);
    if (cgbn_compare(bn_env, hash, threshold) == -1) {
        return true;
    }
    return false;
}

__global__ void kernel_explore(const explore_in_t * __restrict__ explore_params,
                               feistel_state_t<bn_mem_t> *states,
                               explore_out_t * __restrict__ explore_out,
                               uint32_t count)
{
    uint32_t instance = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    if (instance >= count) {
        return;
    }

    context_t bn_context(cgbn_no_checks);
    env_t bn_env(bn_context.env<env_t>());

    bn_t hash, bn_y;
    int64_t index_x = (instance / explore_params->side_length);
    int64_t x = explore_params->x + index_x;
    int64_t y = explore_params->y + (instance % explore_params->side_length);
    coords_to_bn(bn_env, bn_y, y);

    feistel_state_t<bn_t> state{};
    cgbn_load(bn_env, state.l, &(states[index_x].l));
    cgbn_load(bn_env, state.r,  &(states[index_x].r));
    cgbn_set_ui32(bn_env, state.k, explore_params->key);

    inject(bn_env, state, bn_y);
    mix(bn_env, state);
    cgbn_set(bn_env, hash, state.l);

    if (!is_planet(bn_env, hash, explore_params->rarity)) {
        return;
    }

    __shared__ uint32_t result_index[MAX_IPB];
    uint32_t ii = threadIdx.x / TPI;
    uint32_t group_thread = threadIdx.x & TPI-1;
    if (0 == group_thread) {
        result_index[ii] = atomicInc((uint32_t*)&(explore_out->count), 0xffffffff);
    }
    __syncthreads();

    uint32_t i = result_index[ii];
    if (i >= MAX_CUDA_OUT) {
        return;
    }
    explore_out->planets[i].x = x;
    explore_out->planets[i].y = y;
    cgbn_store(bn_env, &(explore_out->planets[i].hash), hash);
}

__global__ void kernel_absorb_x(const explore_in_t * explore_params, feistel_state_t<bn_mem_t> *result, uint32_t count)
{
    uint32_t instance = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    if (instance >= count) {
        return;
    }

    context_t bn_context(cgbn_no_checks);
    env_t bn_env(bn_context.env<env_t>());

    bn_t bn_x;
    int64_t x = explore_params->x + instance;
    coords_to_bn(bn_env, bn_x, x);

    feistel_state_t<bn_t> state{};
    cgbn_set_ui32(bn_env, state.l, 0);
    cgbn_set_ui32(bn_env, state.r, 0);
    cgbn_set_ui32(bn_env, state.k, explore_params->key);
    inject(bn_env, state, bn_x);
    mix(bn_env, state);

    cgbn_store(bn_env, &(result[instance].l), state.l);
    cgbn_store(bn_env, &(result[instance].r), state.r);
}

void init_device_constants(int32_t device_id)
{
    bn_mem_t p;
    bn_mem_t c[MimcConstants::rounds];
    from_mpz(MimcConstants::get_p().get_mpz_t(), p._limbs, BITS / 32);
    for (int32_t i = 0; i < MimcConstants::rounds; ++i) {
        from_mpz(MimcConstants::c_at(i).get_mpz_t(), c[i]._limbs, BITS / 32);
    }
    CUDA_CHECK(hipSetDevice(device_id));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_device_p), &p, sizeof(bn_mem_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_device_c), c, sizeof(bn_mem_t) * MimcConstants::rounds));
}

int32_t get_env_i32(const char *name, int32_t default_value=INT32_MAX)
{
    char *value_str;
    value_str = getenv(name);
    if (value_str == nullptr) {
        return default_value;
    }
    return atoi(value_str);
}

void init() {
    g_tpb = get_env_i32("MIMC_CUDA_BLOCK_SIZE", DEFAULT_TPB);
    if (g_tpb <= 0 || g_tpb > 1024) {
        g_tpb = DEFAULT_TPB;
    }
    int32_t device_id = get_env_i32("MIMC_CUDA_DEVICE", 0);

    printf("cuda block size: %d\n", g_tpb);
    printf("cuda device id: %d\n", device_id);

    init_device_constants(device_id);
}

void get_result(explore_out_t * cuda_result, std::vector<location_hash_t> &hashes)
{
    mpz_class h;
    for (int32_t i = 0; i < cuda_result->count; ++i) {
        to_mpz(h.get_mpz_t(), cuda_result->planets[i].hash._limbs, BITS / 32);
        hashes.push_back({h.get_str(), cuda_result->planets[i].x, cuda_result->planets[i].y});
    }
}

void gpu_explore_chunk(int64_t bottom_left_x,
                       int64_t bottom_left_y,
                       uint32_t side_length,
                       uint32_t key,
                       uint32_t rarity,
                       std::vector<location_hash_t> &hashes)
{
    uint32_t TPB = g_tpb;
    uint32_t IPB = TPB / TPI; // IPB is instances per block

    explore_in_t in_params {
        .x = bottom_left_x,
        .y = bottom_left_y,
        .side_length = side_length,
        .key = key,
        .rarity = rarity
    };

    explore_in_t * gpu_in_params;
    CUDA_CHECK(hipMalloc((void **)&gpu_in_params, sizeof(explore_in_t)));
    CUDA_CHECK(hipMemcpy(gpu_in_params, &in_params, sizeof(explore_in_t), hipMemcpyHostToDevice));

    //count == side_length
    feistel_state_t<bn_mem_t> *states;
    CUDA_CHECK(hipMalloc((void **)&states, sizeof(feistel_state_t<bn_mem_t>) * side_length));
    kernel_absorb_x<<<(side_length + IPB - 1) / IPB, TPB>>>(gpu_in_params, states, side_length);
    CUDA_CHECK(hipDeviceSynchronize());

    explore_out_t *out;
    CUDA_CHECK(hipHostAlloc((void **)&out, sizeof(explore_out_t), hipHostMallocDefault));
    out->count = 0;
    explore_out_t *gpu_out;
    CUDA_CHECK(hipMalloc((void **)&gpu_out, sizeof(explore_out_t)));
    CUDA_CHECK(hipMemcpy(gpu_out, out, sizeof(explore_out_t), hipMemcpyHostToDevice));

    uint32_t count = side_length * side_length;
    kernel_explore<<<(count + IPB - 1) / IPB, TPB>>>(gpu_in_params, states, gpu_out, count);
    CUDA_CHECK(hipDeviceSynchronize());

    // copy the result back from gpuMemory
    CUDA_CHECK(hipMemcpy(out, gpu_out, sizeof(explore_out_t), hipMemcpyDeviceToHost));
    get_result(out, hashes);

    CUDA_CHECK(hipHostFree(out));
    CUDA_CHECK(hipFree(gpu_in_params));
    CUDA_CHECK(hipFree(states));
    CUDA_CHECK(hipFree(gpu_out));
}
